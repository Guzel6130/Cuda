#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/device_functions.h> 
#include ""
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hiprand.h>
#include <math.h>

__global__ void Pi_GPU(float *x, float *y, int *totalCounts, int N) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x; // номер элемента
	int threadCount = gridDim.x * blockDim.x; //cмещение

	int countPoints = 0;
	for (int i = idx; i < N; i += threadCount) {
		if (x[i] * x[i] + y[i] * y[i] < 1) {
			countPoints++;
		}
	}
	atomicAdd(totalCounts, countPoints); // каждый поток суммирует в переменную
}

float PI_CPU(float *x, float *y, int N) {
	int countPoints = 0; //Кол-во точек в круге
	for (int i = 0; i < N; i++) {
		if (x[i] * x[i] + y[i] * y[i] < 1) {
			countPoints++;
		}
	}
	return float(countPoints) * 4 / N;
}

int main(){ 
	// Количество точек 
	const long long N = 20000000;
	// Выделяем память для храния данных на CPU
	float *X, *Y, *devX, *devY;
	X = (float *)calloc(N, sizeof(float));
	Y = (float *)calloc(N, sizeof(float));

	//Выделяем память для храния данных на GPU
	hipMalloc((void **)&devX, N * sizeof(float));
	hipMalloc((void **)&devY, N * sizeof(float));

	//создаем новый генератор
	hiprandGenerator_t curandGenerator; 
	hiprandCreateGenerator(&curandGenerator, HIPRAND_RNG_PSEUDO_DEFAULT); 
	hiprandSetPseudoRandomGeneratorSeed(curandGenerator, 1234ULL); 
	// генерируем числа
	hiprandGenerateUniform(curandGenerator, devX, N); 
	hiprandGenerateUniform(curandGenerator, devY, N);

	hiprandDestroyGenerator(curandGenerator); 

	//Копируем заполненные вектора с GPU на CPU
	hipMemcpy(X, devX, N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(Y, devY, N * sizeof(float), hipMemcpyDeviceToHost);
	
	int blockDim = 512; 
	dim3 threads(blockDim, 1);
	dim3 grid(N / (128 * blockDim), 1);

	int *gpu_total_counts = 0;
	int*gpu_total_counts_host = (int *)calloc(1, sizeof(int));
	hipMalloc((void **)&gpu_total_counts, 512 * sizeof(int));

	//Создаем event'ы для замера времени работы GPU
	float gpuTime = 0;

	hipEvent_t start;
	hipEvent_t stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	//старт расчетов на GPU
	hipEventRecord(start, 0);
	Pi_GPU << <grid, threads >> >(devX, devY, gpu_total_counts, N);
	//Копируем результат с GPU на CPU
	hipMemcpy(gpu_total_counts_host, gpu_total_counts, sizeof(int), hipMemcpyDeviceToHost);
	//число пи на GPU
	float gpu_result = (float) *gpu_total_counts_host * 4 / N;
	//Отмечаем окончание расчета
	hipEventRecord(stop, 0);

	//Синхронизируемя с моментом окончания расчетов
	hipEventSynchronize(stop);

	//Рассчитываем время работы GPU
	hipEventElapsedTime(&gpuTime, start, stop);

	std::cout << "GPU time " << gpuTime << "  Result: " << gpu_result << std::endl;

	//Чистим ресурсы на GPU
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(devX);
	hipFree(devY);
	hipFree(gpu_total_counts);

	clock_t  start_time = clock();
	float cpu_result = PI_CPU(X, Y, N);
	clock_t  end_time = clock();
	std::cout << "CPU time " << (double)((end_time - start_time) * 1000 / CLOCKS_PER_SEC) << "  Result : " << cpu_result << std::endl;

	//Чистим память на CPU
	delete X;
	delete Y;
	return 0;
}
